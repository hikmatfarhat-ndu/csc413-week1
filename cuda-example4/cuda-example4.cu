#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>

__global__ void mmult(float* a, float* b, float* ab, size_t width)
{
    // calculate the row & column index of the element
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float result = 0;

    // do dot product between row of a and column of b
    for (int k = 0; k < width; ++k)
    {
        result += a[row * width + k] * b[k * width + col];
    }

    // write out this thread's result
    ab[row * width + col] = result;
}
__global__ void kernel2(float* da, float* db, float* dc, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float result = 0;
    for (int k = 0; k < width; ++k) 
    {
        result += da[row * width + k] * db[k * width + col];
    }
    dc[row * width + col] = result;
}

void callKernel(float* da, float* db, float* dc, int width, dim3 block_size) {
    dim3 blocksPerGrid(width / block_size.x, width / block_size.y);
    hipEvent_t kernel_start, kernel_end;
    hipEventCreate(&kernel_start);
    hipEventCreate(&kernel_end);
    kernel2 << <blocksPerGrid, block_size >> > (da, db, dc, width);
    float time = 0;
    float total = 0;
    for (int i = 0; i < 100; ++i) {
        hipEventRecord(kernel_start);
        mmult<< <blocksPerGrid, block_size >> > (da, db, dc, width);
        hipEventRecord(kernel_end);
        hipEventSynchronize(kernel_end);
        hipEventElapsedTime(&time, kernel_start, kernel_end);
        total += time;
    }
    std::cout << "time " << total/100 << '\n';
    total = 0.;
    for (int i = 0; i < 100; ++i) {
        hipEventRecord(kernel_start);
        kernel2 << <blocksPerGrid, block_size >> > (da, db, dc, width);
        hipEventRecord(kernel_end);
        hipEventSynchronize(kernel_end);
        hipEventElapsedTime(&time, kernel_start, kernel_end);
        total += time;
    }
    std::cout << "time " << total / 100 << '\n';

}
int main() {
    const int matrix_w = 1024;
    const int msize = matrix_w * matrix_w;
    float* a, * b, * c;

    float* da, * db, * dc;
    a = (float*)malloc(msize * sizeof(float));
    b = (float*)malloc(msize * sizeof(float));
    c = (float*)malloc(msize * sizeof(float));
    for (int i = 0; i < msize; ++i) {
        a[i] = 1;
        b[i] = 1;
    }

    hipMalloc(&da, msize * sizeof(float));
    hipMalloc(&db, msize * sizeof(float));
    hipMalloc(&dc, msize * sizeof(float));
    hipMemcpy(da, a, msize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(db, b, msize * sizeof(float), hipMemcpyHostToDevice);

    /*dim3 threadsPerBlock (matrix_w,matrix_w);
    kernel1<<<1,threadsPerBlock>>>(da,db,dc,matrix_w);*/
    /* total number of threads per block is 1024 which is the maximum */
    dim3 threadsPerBlock(16, 16);
    callKernel(da, db, dc, matrix_w, threadsPerBlock);
    hipMemcpy(c, dc, msize * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < msize; ++i)
        if (c[i] != 1024)std::cout << "ERROR\n";
    //std::cout << c[i] << ' ';
    std::cout << std::endl;
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    free(a);
    free(b);
    free(c);


}