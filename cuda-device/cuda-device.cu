
#include "hip/hip_runtime.h"


#include <iostream>


int main()
{
	int device;

	hipDeviceProp_t properties;
	hipError_t err = hipSuccess;
	err = hipGetDevice(&device);
	err = hipGetDeviceProperties(&properties, device);
	std::cout << "processor count" << properties.multiProcessorCount << std::endl;
	std::cout << "warp size " << properties.warpSize << std::endl;
	std::cout << "name=" << properties.name << std::endl;
	std::cout << "Compute capability " << properties.major << "." << properties.minor << "\n";
	std::cout << "shared Memory/SM " << properties.sharedMemPerMultiprocessor
		<< std::endl;
	//  std::cout<<"max blocks/SM "<<properties.maxBlocksPerMultiProcessor
	 // <<std::endl;
	if (err == hipSuccess)
		printf("device =%d\n", device);
	else
		printf("error getting deivce\n");
	return 0;
}
