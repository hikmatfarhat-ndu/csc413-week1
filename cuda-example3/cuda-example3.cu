#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>


__global__ void kernel(int* a) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	//a[idx] = blockIdx.x;
	a[idx] = sizeof(int);
}

int main() {
	const int N = 1024;
	const int size = N * sizeof(int);
	int* a;
	int* da = 0;
	a = (int*)malloc(size);
	hipMalloc(&da, size);

	kernel << <N / 4, 4 >> > (da);
	hipMemcpy(a, da, size, hipMemcpyDeviceToHost);
	for (int i = 0; i < 30; ++i)
		std::cout << a[i] << ' ';
	std::cout << std::endl;
	hipFree(da);
	free(a);
}