#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>


__global__ void kernel(int* a, int* b, int* c) {
	int idx = threadIdx.x;
	c[idx] = a[idx] + b[idx];
}

int main() {
	int N = 1024;
	int* a, * b, * c;
	int* da, * db, * dc;
	a = (int*)malloc(N * sizeof(int));
	b = (int*)malloc(N * sizeof(int));
	c = (int*)malloc(N * sizeof(int));

	hipMalloc(&da, N * sizeof(int));
	hipMalloc(&db, N * sizeof(int));
	hipMalloc(&dc, N * sizeof(int));
	for (int i = 0; i < N; ++i) {
		a[i] = i;
		b[i] = 2 * i;
	}
	hipMemcpy(da, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(db, b, N * sizeof(int), hipMemcpyHostToDevice);

	kernel << <1, N >> > (da, db, dc);
	hipMemcpy(c, dc, N * sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < 10; ++i)
		std::cout << c[i] << ' ';
	std::cout << std::endl;
	free(a);
	free(b);
	free(c);
	hipFree(db);
	hipFree(dc);
	hipFree(da);

}