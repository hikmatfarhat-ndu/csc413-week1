#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
// https://developer.nvidia.com/blog/six-ways-saxpy/

__global__ void saxpy(int n, float a, float*  x, float*  y,float *z) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)z[i] = a * x[i] + y[i];
}

__global__ void kernel(float* a, float* b, float* c) {
	int idx = threadIdx.x;
	c[idx] = a[idx] + b[idx];
}

int main() {
	int N = 1024;
	float* a, * b, * c;
	float* da, * db, * dc;
	a = (float*)malloc(N * sizeof(float));
	b = (float*)malloc(N * sizeof(float));
	c = (float*)malloc(N * sizeof(float));

	hipMalloc(&da, N * sizeof(float));
	hipMalloc(&db, N * sizeof(float));
	hipMalloc(&dc, N * sizeof(float));
	for (int i = 0; i < N; ++i) {
		a[i] = i;
		b[i] = 2 * i;
	}
	hipMemcpy(da, a, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(db, b, N * sizeof(float), hipMemcpyHostToDevice);

	kernel << <1, N >> > (da, db, dc);
	hipMemcpy(c, dc, N * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < 10; ++i)
		std::cout << c[i] << ' ';
	std::cout << std::endl;
	saxpy << <4, 256 >> > (N, 3.5, da, db, dc);
	hipMemcpy(c, dc, N * sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < 10; ++i)
		std::cout << c[i] << ' ';
	std::cout << std::endl;
	free(a);
	free(b);
	free(c);
	hipFree(db);
	hipFree(dc);
	hipFree(da);

}