#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
//#include "hip/hip_runtime_api.h"
#include <iostream>

__global__ void kernel(int* a) {
	*a = 17;
	
}

int main() {
	int a = 3;
	int* da = 0;
	hipMalloc(&da, sizeof(int));
	kernel << <1, 1 >> > (da);
	hipMemcpy(&a, da, sizeof(int), hipMemcpyDeviceToHost);
	std::cout << a << '\n';
	hipFree(da);

}