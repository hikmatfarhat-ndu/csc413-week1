#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>
#include <iostream>



#define BLOCK_SIZE 32
__global__ void mult(float* da, float* db, float* dc, int width) {

	int by= blockIdx.y;
	int bx = blockIdx.x;
	int ty = threadIdx.y;
	int tx = threadIdx.x;
	int row = by * BLOCK_SIZE + ty;
	int col = bx * BLOCK_SIZE + tx;
	__shared__ float sa[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float sb[BLOCK_SIZE][BLOCK_SIZE];
	float res = 0.0;
	int ntiles = width / BLOCK_SIZE;
	for (int b = 0; b < ntiles; ++b) {
		
		/* copy from memory to shared memory */
		sa[ty][tx] = da[row * width + b * BLOCK_SIZE + tx];
		sb[ty][tx] = db[(b * BLOCK_SIZE + ty) * width + col];
		
		__syncthreads();
		for (int k = 0; k < BLOCK_SIZE; ++k) {
			res += sa[ty][k] * sb[k][tx];
		}
		__syncthreads();
	}
	dc[row* width + col] = res;
}


int main() {
	hipEvent_t kernel_start,kernel_end;
	hipEventCreate(&kernel_start);
	hipEventCreate(&kernel_end);


	float* a, * b, * c;
	float* da, * db, * dc;

	const int matrix_width = 1024;
	const int size = matrix_width * matrix_width;
	a = (float*)malloc(size * sizeof(float));
	b = (float*)malloc(size * sizeof(float));
	c = (float*)malloc(size * sizeof(float));
	for (int i = 0; i < size; ++i) {
		a[i] = 1;
		b[i] = 1;
	}
	hipMalloc(&da, size * sizeof(float));
	hipMalloc(&db, size * sizeof(float));
	hipMalloc(&dc, size * sizeof(float));
	hipMemcpy(da, a, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(db, b, size * sizeof(float), hipMemcpyHostToDevice);
	dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
	dim3 gridSize(matrix_width/ BLOCK_SIZE, matrix_width / BLOCK_SIZE);
	mult << <gridSize, blockSize >> > (da, db, dc, matrix_width);
	float time = 0;
	float total = 0;

	for (int i = 0; i < 500; ++i) {
		hipEventRecord(kernel_start,0);
		mult << <gridSize, blockSize >> > (da, db, dc, matrix_width);
		hipEventRecord(kernel_end,0);
		hipEventSynchronize(kernel_end);
		hipEventElapsedTime(&time, kernel_start, kernel_end);
		total += time;
	}
	std::cout << "average time " << total/500 << '\n';
	hipMemcpy(c, dc, size * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < 64; i++)
		std::cout << c[i] << ' ';
	std::cout << std::endl;
	hipFree(da);
	hipFree(db);
	hipFree(dc);
	free(a);
	free(b);
	free(c);

}