#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


#define BLOCK_SIZE 32
__global__ void mult(float* da, float* db, float* dc, int width) {

	int by= blockIdx.y;
	int bx = blockIdx.x;
	int ty = threadIdx.y;
	int tx = threadIdx.x;
	//int i = BLOCK_SIZE * brow + row;
	//int j = BLOCK_SIZE * bcol + col;
	float res = 0.0;
	for (int b = 0; b < width / BLOCK_SIZE; ++b) {
		__shared__ float sa[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float sb[BLOCK_SIZE][BLOCK_SIZE];
		/* copy from memory to shared memory */
		sa[ty][tx] = da[(by * BLOCK_SIZE + ty) * width + b * BLOCK_SIZE + tx];
		sb[ty][tx] = db[(b * BLOCK_SIZE + ty) * width + bx * BLOCK_SIZE + tx];
		
		__syncthreads();
		for (int k = 0; k < BLOCK_SIZE; ++k) {
			res += sa[ty][k] * sb[k][tx];

		}
		__syncthreads();
	}
	//dc[(by * BLOCK_SIZE + ty)* width + bx * BLOCK_SIZE + tx] = res;
	dc[(by * BLOCK_SIZE * width + bx * BLOCK_SIZE) + width * ty + tx] = res;
}


int main() {
	hipEvent_t kernel_start,kernel_end;
	hipEventCreate(&kernel_start);
	hipEventCreate(&kernel_end);


	float* a, * b, * c;
	float* da, * db, * dc;

	const int matrix_width = 1024;
	const int size = matrix_width * matrix_width;
	a = (float*)malloc(size * sizeof(float));
	b = (float*)malloc(size * sizeof(float));
	c = (float*)malloc(size * sizeof(float));
	for (int i = 0; i < size; ++i) {
		a[i] = 1;
		b[i] = 1;
	}
	hipMalloc(&da, size * sizeof(float));
	hipMalloc(&db, size * sizeof(float));
	hipMalloc(&dc, size * sizeof(float));
	hipMemcpy(da, a, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(db, b, size * sizeof(float), hipMemcpyHostToDevice);
	dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
	dim3 gridSize(matrix_width/ BLOCK_SIZE, matrix_width / BLOCK_SIZE);
	mult << <gridSize, blockSize >> > (da, db, dc, matrix_width);
	float time = 0;
	float total = 0;

	for (int i = 0; i < 100; ++i) {
		hipEventRecord(kernel_start);
		mult << <gridSize, blockSize >> > (da, db, dc, matrix_width);
		hipEventRecord(kernel_end);
		hipEventSynchronize(kernel_end);
		hipEventElapsedTime(&time, kernel_start, kernel_end);
		total += time;
	}
	std::cout << "average time " << total/100 << '\n';
	hipMemcpy(c, dc, size * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < 64; i++)
		std::cout << c[i] << ' ';
	std::cout << std::endl;
	hipFree(da);
	hipFree(db);
	hipFree(dc);
	free(a);
	free(b);
	free(c);

}